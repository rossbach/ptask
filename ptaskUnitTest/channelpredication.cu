
#include <hip/hip_runtime.h>
// Device code
extern "C" __global__ void scale(float* A, float * B, float scalar, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        A[i] = A[i]*scalar;
        B[i] = B[i]*scalar;
    }
}
