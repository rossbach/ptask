
#include <hip/hip_runtime.h>
// Device code
extern "C" __global__ void op(float* A, float scalar, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {
        A[i] = A[i]*scalar;
    }
}
